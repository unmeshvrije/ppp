// CUDA Device Query

#include <stdio.h>

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
	printf("Name : %s\n", devProp.name);
	printf("Major revision number:         %d\n",  devProp.major);
	printf("Minor revision number:         %d\n",  devProp.minor);
	printf("Name:                          %s\n",  devProp.name);
	printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
	printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
	printf("Warp size:                     %d\n",  devProp.warpSize);
	printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
	printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %u\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %u\n", i, devProp.maxGridSize[i]);
	printf("Clock rate:                    %d\n",  devProp.clockRate);
	printf("Total constant memory:         %u\n",  devProp.totalConstMem);
	printf("Texture alignment:             %u\n",  devProp.textureAlignment);
	printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
	printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
	printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	return;
}

int main()
{
	// Number of CUDA devices
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("CUDA Device Query...\n");
	printf("There are %d CUDA devices.\n", devCount);

	// Iterate through devices
	devCount = 4;
	for (int i = 0; i < devCount; ++i)
	{
		hipSetDevice(i);
		// Get device properties
		printf("\nCUDA Device #%d\n", i);
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		printDevProp(devProp);
	}

	printf("\nPress any key to exit...");
	char c;
	scanf("%c", &c);

	return 0;
}
